#include "hip/hip_runtime.h"
#include "rsmpl.h"

// void HandleError( hipError_t err, const char *file, int line ){
//     if (err != hipSuccess) {
//         printf( "%s in %s at line %d\n", hipGetErrorString( err ), file, line );
//         exit( EXIT_FAILURE );
//     }
// }
//..................................................................................................................................


__constant__ float cA[12];

__global__
void d_rsmpl(float *imr,
	const float *imo,
	Cimg Cim) {

	// extern __shared__ float s[];

	int ib = blockIdx.x + blockIdx.y*gridDim.x + blockIdx.z*gridDim.x*gridDim.y;
	//int it = threadIdx.x + threadIdx.y*blockDim.x + threadIdx.z*blockDim.x*blockDim.y;


	float x = (blockIdx.x*Cim.VXSOx + Cim.OFFOx) + Cim.VXSOx / VDIV*(0.5 + threadIdx.x);
	float y = -(blockIdx.y*Cim.VXSOy + Cim.OFFOy) - Cim.VXSOy / VDIV*(0.5 + threadIdx.y);
	float z = blockIdx.z*Cim.VXSOz + Cim.OFFOz + Cim.VXSOz / VDIV*(0.5 + threadIdx.z);

	float xp = cA[0] * x + cA[1] * y + cA[2] * z + cA[3];
	float yp = cA[4] * x + cA[5] * y + cA[6] * z + cA[7];
	float zp = cA[8] * x + cA[9] * y + cA[10] * z + cA[11];

	short u = roundf(-Cim.OFFRx / Cim.VXSRx) + floorf((xp) / Cim.VXSRx);
	short v = roundf(-Cim.OFFRy / Cim.VXSRy) - ceilf((yp) / Cim.VXSRy);
	short w = roundf(-Cim.OFFRz / Cim.VXSRz) + floorf((zp) / Cim.VXSRz);

	if ((u<Cim.VXNRx) && (v<Cim.VXNRy) && (w<Cim.VXNRz) && (u >= 0) && (v >= 0) && (w >= 0))
		atomicAdd(imr + u + v*Cim.VXNRx + w*Cim.VXNRx*Cim.VXNRy, imo[ib] / (VDIV*VDIV*VDIV));

}

//::::::::::::::::::::::::::::::::::::::::::::::::::::::::::::::::::::::::::::::::::::::::::::::::::::::::::::::::::::::::::::::::::


float *rsmpl(float *imo,
	float *A,
	Cimg Cim)

{

	float *d_imr;
	HANDLE_ERROR(hipMalloc(&d_imr, Cim.VXNRx*Cim.VXNRy*Cim.VXNRz * sizeof(float)));
	HANDLE_ERROR(hipMemset(d_imr, 0, Cim.VXNRx*Cim.VXNRy*Cim.VXNRz * sizeof(float)));

	float *d_imo;
	HANDLE_ERROR(hipMalloc(&d_imo, Cim.VXNOx*Cim.VXNOy*Cim.VXNOz * sizeof(float)));
	HANDLE_ERROR(hipMemcpy(d_imo, imo, Cim.VXNOx*Cim.VXNOy*Cim.VXNOz * sizeof(float), hipMemcpyHostToDevice));


	hipMemcpyToSymbol(HIP_SYMBOL(cA), A, 12 * sizeof(float));
	// double * d_A;
	// HANDLE_ERROR( hipMalloc(&d_A, 12*sizeof(double)) );
	// HANDLE_ERROR( hipMemcpy(d_A, A, 12*sizeof(double), hipMemcpyHostToDevice) );


	//<><><><><><><><><><><><><><><><><><><><><><><><><><><><><><><><><><><><><><><><><><><><>
	printf("i> calculating transformation with %d samples per voxel...", VDIV);
	dim3 grid(Cim.VXNOx, Cim.VXNOy, Cim.VXNOz);
	dim3 block(VDIV, VDIV, VDIV);
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, 0);
	d_rsmpl << <grid, block >> >(d_imr, d_imo, Cim);
	hipError_t error = hipGetLastError();
	if (error != hipSuccess) { printf("CUDA kernel for image resampling: error: %s\n", hipGetErrorString(error)); exit(-1); }
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	float elapsedTime;
	hipEventElapsedTime(&elapsedTime, start, stop);
	hipEventDestroy(start);
	hipEventDestroy(stop);
	printf("DONE in %fs.\n\n", 0.001*elapsedTime);
	//<><><><><><><><><><><><><><><><><><><><><><><><><><><><><><><><><><><><><><><><><><><><>


	//allocate memory for the resampled image for the return
	float *imr = (float*)malloc(Cim.VXNRx*Cim.VXNRy*Cim.VXNRz * sizeof(float));
	//copy the image from GPU to CPU
	HANDLE_ERROR(hipMemcpy(imr, d_imr, Cim.VXNRx*Cim.VXNRy*Cim.VXNRz * sizeof(float), hipMemcpyDeviceToHost));


	return imr;
}


