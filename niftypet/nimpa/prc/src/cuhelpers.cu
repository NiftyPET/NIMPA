#include <cstdio>
#include "cuhelpers.h"

void HandleError(hipError_t err, const char *file, int line) {
	if (err != hipSuccess) {
		printf("%s in %s at line %d\n", hipGetErrorString(err), file, line);
		exit(EXIT_FAILURE);
	}
}
